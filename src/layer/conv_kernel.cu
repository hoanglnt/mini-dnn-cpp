#include "hip/hip_runtime.h"
#include <iostream>
#include "conv_kernel.h"
#define TILE_SIZE 16
__constant__ float const_weights[2560];

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

static GpuTimer timer;
void startTimer()
{
    timer.Start();
}

float stopTimer()
{
    timer.Stop();

	return timer.Elapsed();
}   

__global__ void matrix_multiplication_kernel(float* A, float* C, int m, int n, int k) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    float sum = 0;

    for (int ph = 0; ph < ceil(n / (float)TILE_SIZE); ++ph) {
        if (row < m && ph * TILE_SIZE + tx < n)
            As[ty][tx] = A[row * n + ph * TILE_SIZE + tx];
        else
            As[ty][tx] = 0;

        if (col < k && ph * TILE_SIZE + ty < n)
            Bs[ty][tx] = const_weights[(ph * TILE_SIZE + ty) * k + col];
        else
            Bs[ty][tx] = 0;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += As[ty][k] * Bs[k][tx];

        __syncthreads();
    }

    if (row < m && col < k)
        C[row * k + col] = sum;
}


void copyWeightsToConstant(float* host_weights, size_t num_weights) {
    hipMemcpyToSymbol(HIP_SYMBOL(const_weights), host_weights, sizeof(float) * num_weights);
}

void im2col_gpu(const float* image, float* data_col, int height_in, int width_in, int channel_in, int height_out, int width_out, int height_kernel, int width_kernel, int pad_h, int pad_w, int stride) {
    // Allocate memory on device
    float *d_image, *d_data_col;
    size_t image_size = sizeof(float) * height_in * width_in * channel_in;
    size_t data_col_size = sizeof(float) * height_out * width_out * height_kernel * width_kernel * channel_in;
    hipMalloc(&d_image, image_size);
    hipMalloc(&d_data_col, data_col_size);

    // Copy image to device
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    int threads = 256; // This can be tuned for your specific GPU
    int blocks = (height_out * width_out + threads - 1) / threads;

    // Launch kernel    
    im2col_kernel<<<blocks, threads>>>(d_image, d_data_col, height_in, width_in, channel_in, height_out, width_out, height_kernel, width_kernel, pad_h, pad_w, stride);

    // Copy result back to host
    hipMemcpy(data_col, d_data_col, data_col_size, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_image);
    hipFree(d_data_col);
}

void matrix_multiply_gpu2(const float* A, float* C, int m, int n, int k) {
    // Allocate device memory for A and C only
    float *d_A, *d_C;
    hipMalloc(&d_A, sizeof(float) * m * n);
    hipMalloc(&d_C, sizeof(float) * m * k);

    // Copy host memory to device for A
    hipMemcpy(d_A, A, sizeof(float) * m * n, hipMemcpyHostToDevice);

    // Kernel launch parameters
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE); // This can be tuned
    dim3 numBlocks((k + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the modified matrix multiplication kernel
    matrix_multiplication_kernel2<<<numBlocks, threadsPerBlock>>>(d_A, d_C, m, n, k);

    // Copy result back to host
    hipMemcpy(C, d_C, sizeof(float) * m * k, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_C);
}

__global__ void im2col_kernel1(float* image, float* data_col, int height_in, int width_in, int channel_in, int height_out, int width_out, int height_kernel, int width_kernel, int pad_h, int pad_w, int stride) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= height_out * width_out) return; // One thread per output element

    int h_out = index / width_out; // Determine which output row and column this thread should handle
    int w_out = index % width_out;

    for (int c = 0; c < channel_in; c++) { // For each channel
        for (int i = 0; i < height_kernel; i++) { // For each row in the kernel
            for (int j = 0; j < width_kernel; j++) { // For each column in the kernel

                int im_row = h_out * stride - pad_h + i; // Calculate corresponding input row
                int im_col = w_out * stride - pad_w + j; // Calculate corresponding input column

                float val = 0; // Default to zero for padding
                if (im_row >= 0 && im_row < height_in && im_col >= 0 && im_col < width_in) {
                    val = image[im_row * width_in + im_col + c * height_in * width_in]; // Adjust for channel
                }

                // Calculate index in data_col
                int data_col_idx = (c * height_kernel * width_kernel + i * width_kernel + j) * height_out * width_out + index;
                data_col[data_col_idx] = val;
            }
        }
    }
}

void im2col_gpu1(const float* image, float* data_col, int height_in, int width_in, int channel_in, int height_out, int width_out, int height_kernel, int width_kernel, int pad_h, int pad_w, int stride) {
    // Allocate memory on device
    float *d_image, *d_data_col;
    size_t image_size = sizeof(float) * height_in * width_in * channel_in;
    size_t data_col_size = sizeof(float) * height_out * width_out * height_kernel * width_kernel * channel_in;
    hipMalloc(&d_image, image_size);
    hipMalloc(&d_data_col, data_col_size);

    // Copy image to device
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    int threads = 256; // This can be tuned for your specific GPU
    int blocks = (height_out * width_out + threads - 1) / threads;

    // Launch kernel
    im2col_kernel1<<<blocks, threads>>>(d_image, d_data_col, height_in, width_in, channel_in, height_out, width_out, height_kernel, width_kernel, pad_h, pad_w, stride);

    // Copy result back to host
    hipMemcpy(data_col, d_data_col, data_col_size, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_image);
    hipFree(d_data_col);
}


__global__ void matrix_multiplication_kernel1(float* A, float* B, float* C, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= m || col >= k) return;
    float t = 0;
    for (int h = 0; h < n; h++)
        t += A[row*n+h] * B[h*k+col];
    C[row*k+col] = t;
}

void matrix_multiply_gpu1(const float* A, const float* B, float* C, int m, int n, int k) {
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(float) * m * n);
    hipMalloc(&d_B, sizeof(float) * n * k);
    hipMalloc(&d_C, sizeof(float) * m * k);

    // Copy host memory to device
    hipMemcpy(d_A, A, sizeof(float) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * n * k, hipMemcpyHostToDevice);

    // Kernel launch parameters
    dim3 threadsPerBlock(16, 16); // This can be tuned
    dim3 numBlocks((k + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the matrix multiplication kernel
    matrix_multiplication_kernel1<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);

    // Copy result back to host
    hipMemcpy(C, d_C, sizeof(float) * m * k, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}