#include "hip/hip_runtime.h"
#include <iostream>
#include "conv_kernel.h"

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
			exit(EXIT_FAILURE);                                    \
		}                                                          \
	}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

static GpuTimer timer;
void startTimer()
{
    timer.Start();
}

float stopTimer()
{
    timer.Stop();

	return timer.Elapsed();
}

__global__ void unrollKernel_1(int C, int H, int W, int K, float* image, float* data_col)
{
	int c, s, h_out, w_out, h_unroll, w_unroll, w_base, p, q;
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	int W_unroll = H_out * W_out;

	if (t < C * W_unroll)
	{
		c = t / W_unroll;
		s = t % W_unroll;
		h_out = s / W_out;
		w_out = s % W_out;
		h_unroll = h_out * W_out + w_out;
		w_base = c * (K * K);

		for (p = 0; p < K; p++)
		{
			for (q = 0; q < K; q++)
			{
				w_unroll = w_base + p * K + q;
				data_col[w_unroll * W_unroll + h_unroll] = image[c * H * W + (h_out + p) * W + (w_out + q)];
			}
		}
	}
}

void unrollGPUWrapper(int C, int H, int W, int K, float* image, float* data_col)
{
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	int W_unroll = H_out * W_out;
	int num_threads = C * H_out * W_out;
	int block_size = 1024;
	int num_blocks = ceil((float)num_threads / block_size);
	
	// Copy image to device
	float* d_image;
	CHECK(hipMalloc(&d_image, C * H * W * sizeof(float)));
	CHECK(hipMemcpy(d_image, image, C * H * W * sizeof(float), hipMemcpyHostToDevice));

	// Copy data_col to device
	float* d_data_col;
	CHECK(hipMalloc(&d_data_col, C * K * K * W_unroll * sizeof(float)));

	unrollKernel_1<<<num_blocks, block_size>>>(C, H, W, K, d_image, d_data_col);
	CHECK(hipGetLastError());

	// Copy data_col back to host
	CHECK(hipMemcpy(data_col, d_data_col, C * K * K * W_unroll * sizeof(float), hipMemcpyDeviceToHost));
	// Free memory
	CHECK(hipFree(d_image));
	CHECK(hipFree(d_data_col));
}